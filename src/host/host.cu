#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../kernel.cuh"
#include "../utils.cuh"

void convertRgb2Gray(uchar3 *inPixels, int width, int height, int *out)
{
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            int r = inPixels[i].x;
            int g = inPixels[i].y;
            int b = inPixels[i].z;
            out[i] = int(0.299f * r + 0.587f * g + 0.114f * b);
            ;
        }
    }
}

void removeSeam(uchar3 *inPixels, int width, int height, int seamIdx, int *path, uchar3 *outPixels)
{
    int delimIdx = seamIdx;
    copyRow(inPixels, width, height, delimIdx, 0, outPixels);

    for (int i = 1; i < height; i++)
    {
        delimIdx = path[(i - 1) * width + delimIdx];
        copyRow(inPixels, width, height, delimIdx, i, outPixels);
    }
}

void calConvolution(int *grayPixels, int width, int height, float *filter, int filterWidth, int *outPixels)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int idx_1d = y * width + x;
            int ele = 0;

            for (int dy = -filterWidth / 2; dy <= filterWidth / 2; dy++)
            {
                for (int dx = -filterWidth / 2; dx <= filterWidth / 2; dx++)
                {
                    int conv_x = max(min(x + dx, width - 1), 0);
                    int conv_y = max(min(y + dy, height - 1), 0);

                    int filter_x = dx + filterWidth / 2;
                    int filter_y = dy + filterWidth / 2;
                    float ele_conv = filter[filter_y * filterWidth + filter_x];

                    ele += int(grayPixels[conv_y * width + conv_x] * ele_conv);
                }
            }

            outPixels[idx_1d] = (int)ele;
        }
    }
}

void calEnergies(int *gx, int *gy, int width, int height, int *energies)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int idx = width * y + x;
            energies[idx] = sqrt(gx[idx] * gx[idx] + gy[idx] * gy[idx]);
        }
    }
}

void applySeamCarving(uchar3 *inPixels, int width, int height, int nSeams, uchar3 *&outPixels)
{
    uchar3 *src = inPixels;
    uchar3 *out = (uchar3 *)malloc((width - 1) * height * sizeof(uchar3));
    ;
    // int outHeight = height;
    int srcWidth = width, srcHeight = height;
    float *gx, *gy;
    createSobelFilters(gx, gy);

    float *gaussFilter;
    createGaussianFilter(gaussFilter);

    for (int i = 1; i <= nSeams; i++)
    {
        int outWidth = width - i;
        if (i > 1)
        {
            out = (uchar3 *)realloc(out, outWidth * height * sizeof(uchar3));
        }

        // 1. Convert img to grayscale
        int *grayscaleImg = (int *)malloc(srcWidth * srcHeight * sizeof(int));
        convertRgb2Gray(src, srcWidth, srcHeight, grayscaleImg);

        // 2. Calculate energy value for each pixels: blur --> dx, dy --> energy = |dx| + |dy|
        int *blurImg = (int *)malloc(srcWidth * srcHeight * sizeof(int));
        calConvolution(grayscaleImg, srcWidth, srcHeight, gaussFilter, BLUR_KERNEL_SIZE, blurImg);

        int *dx = (int *)malloc(srcWidth * srcHeight * sizeof(int));
        int *dy = (int *)malloc(srcWidth * srcHeight * sizeof(int));

        calConvolution(blurImg, srcWidth, srcHeight, gx, SOBEL_KERNEL_SIZE, dx);
        calConvolution(blurImg, srcWidth, srcHeight, gy, SOBEL_KERNEL_SIZE, dy);

        int *energy = (int *)malloc(srcWidth * srcHeight * sizeof(int));
        calEnergies(dx, dy, srcWidth, srcHeight, energy);

        // 3. Find seam given energy values above

        int *path = (int *)malloc(srcWidth * srcHeight * sizeof(int));
        int seamIdx = -1;
        findSeam(energy, srcWidth, srcHeight, seamIdx, path);

        // 4. Remove seam
        removeSeam(src, srcWidth, srcHeight, seamIdx, path, out);

        // 5. Reassign variables for next iteration
        src = out;
        srcWidth--;

        free(grayscaleImg);
        free(blurImg);
        free(energy);
        free(path);
        free(dx);
        free(dy);
    }

    outPixels = out;

    // Free allocated memory
    free(gx);
    free(gy);
    free(gaussFilter);
}

int main(int argc, char ** argv) {
    GpuTimer timer;

    int width, height;
    uchar3 *inPixels = NULL;
    uchar3 *outPixels = NULL;

    readPnm(argv[1], width, height, inPixels);
    char * outFileNameBase = strtok(argv[2], ".");
    int nSeams = atoi(argv[3]);

    for (int i = 0; i < 1; i++)
    {

        timer.Start();
        applySeamCarving(inPixels, width, height, nSeams, outPixels);
        timer.Stop();

        printf("Version CPU, %d seams: %f ms\n", nSeams, timer.Elapsed());
        writePnm(outPixels, width - nSeams, height, concatStr(outFileNameBase, "_host.pnm"));
    }
}